#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/gpu/device/functional.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/transform.hpp"
#include "opencv2/gpu/device/limits.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"
#include "internal_shared.hpp"

using namespace cv::gpu;
using namespace cv::gpu::device;

namespace cv { namespace gpu { namespace mathfunc
{

    //////////////////////////////////////////////////////////////////////////////////////
    // Compare

    template <typename T1, typename T2> struct NotEqual : binary_function<T1, T2, uchar>
    {
        __device__ __forceinline__ uchar operator()(const T1& src1, const T2& src2) const
        {
            return static_cast<uchar>(static_cast<int>(src1 != src2) * 255);
        }
    };

    template <typename T1, typename T2>
    inline void compare_ne(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst, hipStream_t stream)
    {
        NotEqual<T1, T2> op;
        transform(static_cast< DevMem2D_<T1> >(src1), static_cast< DevMem2D_<T2> >(src2), dst, op, stream);
    }

    void compare_ne_8uc4(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst, hipStream_t stream)
    {
        compare_ne<uint, uint>(src1, src2, dst, stream);
    }
    void compare_ne_32f(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst, hipStream_t stream)
    {
        compare_ne<float, float>(src1, src2, dst, stream);
    }


    //////////////////////////////////////////////////////////////////////////
    // Unary bitwise logical matrix operations

    enum { UN_OP_NOT };

    template <typename T, int opid>
    struct UnOp;

    template <typename T>
    struct UnOp<T, UN_OP_NOT>
    { 
        static __device__ __forceinline__ T call(T v) { return ~v; }
    };


    template <int opid>
    __global__ void bitwiseUnOpKernel(int rows, int width, const PtrStep src, PtrStep dst)
    {
        const int x = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (y < rows) 
        {
            uchar* dst_ptr = dst.ptr(y) + x;
            const uchar* src_ptr = src.ptr(y) + x;
            if (x + sizeof(uint) - 1 < width)
            {
                *(uint*)dst_ptr = UnOp<uint, opid>::call(*(uint*)src_ptr);
            }
            else
            {
                const uchar* src_end = src.ptr(y) + width;
                while (src_ptr < src_end)
                {
                    *dst_ptr++ = UnOp<uchar, opid>::call(*src_ptr++);
                }
            }
        }
    }


    template <int opid>
    void bitwiseUnOp(int rows, int width, const PtrStep src, PtrStep dst, 
                     hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(width, threads.x * sizeof(uint)), 
                  divUp(rows, threads.y));

        bitwiseUnOpKernel<opid><<<grid, threads>>>(rows, width, src, dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0) 
            cudaSafeCall( hipDeviceSynchronize() );
    }


    template <typename T, int opid>
    __global__ void bitwiseUnOpKernel(int rows, int cols, int cn, const PtrStep src, 
                                      const PtrStep mask, PtrStep dst)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows && mask.ptr(y)[x / cn]) 
        {
            T* dst_row = (T*)dst.ptr(y);
            const T* src_row = (const T*)src.ptr(y);

            dst_row[x] = UnOp<T, opid>::call(src_row[x]);
        }
    }


    template <typename T, int opid>
    void bitwiseUnOp(int rows, int cols, int cn, const PtrStep src, 
                     const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        bitwiseUnOpKernel<T, opid><<<grid, threads>>>(rows, cols, cn, src, mask, dst); 
        cudaSafeCall( hipGetLastError() );

        if (stream == 0) 
            cudaSafeCall( hipDeviceSynchronize() );
    }


    void bitwiseNotCaller(int rows, int cols, size_t elem_size1, int cn, 
                          const PtrStep src, PtrStep dst, hipStream_t stream)
    {
        bitwiseUnOp<UN_OP_NOT>(rows, static_cast<int>(cols * elem_size1 * cn), src, dst, stream);
    }


    template <typename T>
    void bitwiseMaskNotCaller(int rows, int cols, int cn, const PtrStep src, 
                              const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        bitwiseUnOp<T, UN_OP_NOT>(rows, cols * cn, cn, src, mask, dst, stream);
    }

    template void bitwiseMaskNotCaller<uchar>(int, int, int, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskNotCaller<ushort>(int, int, int, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskNotCaller<uint>(int, int, int, const PtrStep, const PtrStep, PtrStep, hipStream_t);


    //////////////////////////////////////////////////////////////////////////
    // Binary bitwise logical matrix operations

    enum { BIN_OP_OR, BIN_OP_AND, BIN_OP_XOR };

    template <typename T, int opid>
    struct BinOp;

    template <typename T>
    struct BinOp<T, BIN_OP_OR>
    { 
        static __device__ __forceinline__ T call(T a, T b) { return a | b; } 
    };


    template <typename T>
    struct BinOp<T, BIN_OP_AND>
    { 
        static __device__ __forceinline__ T call(T a, T b) { return a & b; } 
    };

    template <typename T>
    struct BinOp<T, BIN_OP_XOR>
    { 
        static __device__ __forceinline__ T call(T a, T b) { return a ^ b; } 
    };


    template <int opid>
    __global__ void bitwiseBinOpKernel(int rows, int width, const PtrStep src1, 
                                       const PtrStep src2, PtrStep dst)
    {
        const int x = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (y < rows) 
        {
            uchar* dst_ptr = dst.ptr(y) + x;
            const uchar* src1_ptr = src1.ptr(y) + x;
            const uchar* src2_ptr = src2.ptr(y) + x;

            if (x + sizeof(uint) - 1 < width)
            {
                *(uint*)dst_ptr = BinOp<uint, opid>::call(*(uint*)src1_ptr, *(uint*)src2_ptr);
            }
            else
            {
                const uchar* src1_end = src1.ptr(y) + width;
                while (src1_ptr < src1_end)
                {
                    *dst_ptr++ = BinOp<uchar, opid>::call(*src1_ptr++, *src2_ptr++);
                }
            }
        }
    }


    template <int opid>
    void bitwiseBinOp(int rows, int width, const PtrStep src1, const PtrStep src2, 
                      PtrStep dst, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(width, threads.x * sizeof(uint)), divUp(rows, threads.y));

        bitwiseBinOpKernel<opid><<<grid, threads>>>(rows, width, src1, src2, dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0) 
            cudaSafeCall( hipDeviceSynchronize() );
    }


    template <typename T, int opid>
    __global__ void bitwiseBinOpKernel(
            int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
            const PtrStep mask, PtrStep dst)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows && mask.ptr(y)[x / cn]) 
        {
            T* dst_row = (T*)dst.ptr(y);
            const T* src1_row = (const T*)src1.ptr(y);
            const T* src2_row = (const T*)src2.ptr(y);

            dst_row[x] = BinOp<T, opid>::call(src1_row[x], src2_row[x]);
        }
    }


    template <typename T, int opid>
    void bitwiseBinOp(int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
                        const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        bitwiseBinOpKernel<T, opid><<<grid, threads>>>(rows, cols, cn, src1, src2, mask, dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0) 
            cudaSafeCall( hipDeviceSynchronize() );
    }


    void bitwiseOrCaller(int rows, int cols, size_t elem_size1, int cn, const PtrStep src1, 
                         const PtrStep src2, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<BIN_OP_OR>(rows, static_cast<int>(cols * elem_size1 * cn), src1, src2, dst, stream);
    }


    template <typename T>
    void bitwiseMaskOrCaller(int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
                             const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<T, BIN_OP_OR>(rows, cols * cn, cn, src1, src2, mask, dst, stream);
    }

    template void bitwiseMaskOrCaller<uchar>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskOrCaller<ushort>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskOrCaller<uint>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);


    void bitwiseAndCaller(int rows, int cols, size_t elem_size1, int cn, const PtrStep src1, 
                          const PtrStep src2, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<BIN_OP_AND>(rows, static_cast<int>(cols * elem_size1 * cn), src1, src2, dst, stream);
    }


    template <typename T>
    void bitwiseMaskAndCaller(int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
                              const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<T, BIN_OP_AND>(rows, cols * cn, cn, src1, src2, mask, dst, stream);
    }

    template void bitwiseMaskAndCaller<uchar>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskAndCaller<ushort>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskAndCaller<uint>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);


    void bitwiseXorCaller(int rows, int cols, size_t elem_size1, int cn, const PtrStep src1, 
                          const PtrStep src2, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<BIN_OP_XOR>(rows, static_cast<int>(cols * elem_size1 * cn), src1, src2, dst, stream);
    }


    template <typename T>
    void bitwiseMaskXorCaller(int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
                              const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<T, BIN_OP_XOR>(rows, cols * cn, cn, src1, src2, mask, dst, stream);
    }

    template void bitwiseMaskXorCaller<uchar>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskXorCaller<ushort>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskXorCaller<uint>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);


    //////////////////////////////////////////////////////////////////////////
    // min/max
    
    template <typename T>
    void min_gpu(const DevMem2D_<T>& src1, const DevMem2D_<T>& src2, const DevMem2D_<T>& dst, hipStream_t stream)
    {
        transform(src1, src2, dst, minimum<T>(), stream);    
    }

    template void min_gpu<uchar >(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst, hipStream_t stream);
    template void min_gpu<schar >(const DevMem2D_<schar>& src1, const DevMem2D_<schar>& src2, const DevMem2D_<schar>& dst, hipStream_t stream);
    template void min_gpu<ushort>(const DevMem2D_<ushort>& src1, const DevMem2D_<ushort>& src2, const DevMem2D_<ushort>& dst, hipStream_t stream);
    template void min_gpu<short >(const DevMem2D_<short>& src1, const DevMem2D_<short>& src2, const DevMem2D_<short>& dst, hipStream_t stream);
    template void min_gpu<int   >(const DevMem2D_<int>& src1, const DevMem2D_<int>& src2, const DevMem2D_<int>& dst, hipStream_t stream);
    template void min_gpu<float >(const DevMem2D_<float>& src1, const DevMem2D_<float>& src2, const DevMem2D_<float>& dst, hipStream_t stream);
    template void min_gpu<double>(const DevMem2D_<double>& src1, const DevMem2D_<double>& src2, const DevMem2D_<double>& dst, hipStream_t stream);

    template <typename T>
    void max_gpu(const DevMem2D_<T>& src1, const DevMem2D_<T>& src2, const DevMem2D_<T>& dst, hipStream_t stream)
    {
        transform(src1, src2, dst, maximum<T>(), stream);    
    }
    
    template void max_gpu<uchar >(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst, hipStream_t stream);
    template void max_gpu<schar >(const DevMem2D_<schar>& src1, const DevMem2D_<schar>& src2, const DevMem2D_<schar>& dst, hipStream_t stream);
    template void max_gpu<ushort>(const DevMem2D_<ushort>& src1, const DevMem2D_<ushort>& src2, const DevMem2D_<ushort>& dst, hipStream_t stream);
    template void max_gpu<short >(const DevMem2D_<short>& src1, const DevMem2D_<short>& src2, const DevMem2D_<short>& dst, hipStream_t stream);
    template void max_gpu<int   >(const DevMem2D_<int>& src1, const DevMem2D_<int>& src2, const DevMem2D_<int>& dst, hipStream_t stream);
    template void max_gpu<float >(const DevMem2D_<float>& src1, const DevMem2D_<float>& src2, const DevMem2D_<float>& dst, hipStream_t stream);
    template void max_gpu<double>(const DevMem2D_<double>& src1, const DevMem2D_<double>& src2, const DevMem2D_<double>& dst, hipStream_t stream);

    template <typename T>
    void min_gpu(const DevMem2D_<T>& src1, T src2, const DevMem2D_<T>& dst, hipStream_t stream)
    {
        transform(src1, dst, device::bind2nd(minimum<T>(), src2), stream);    
    }

    template void min_gpu<uchar >(const DevMem2D& src1, uchar src2, const DevMem2D& dst, hipStream_t stream);
    template void min_gpu<schar >(const DevMem2D_<schar>& src1, schar src2, const DevMem2D_<schar>& dst, hipStream_t stream);
    template void min_gpu<ushort>(const DevMem2D_<ushort>& src1, ushort src2, const DevMem2D_<ushort>& dst, hipStream_t stream);
    template void min_gpu<short >(const DevMem2D_<short>& src1, short src2, const DevMem2D_<short>& dst, hipStream_t stream);
    template void min_gpu<int   >(const DevMem2D_<int>& src1, int src2, const DevMem2D_<int>& dst, hipStream_t stream);
    template void min_gpu<float >(const DevMem2D_<float>& src1, float src2, const DevMem2D_<float>& dst, hipStream_t stream);
    template void min_gpu<double>(const DevMem2D_<double>& src1, double src2, const DevMem2D_<double>& dst, hipStream_t stream);

    template <typename T>
    void max_gpu(const DevMem2D_<T>& src1, T src2, const DevMem2D_<T>& dst, hipStream_t stream)
    {
        transform(src1, dst, device::bind2nd(maximum<T>(), src2), stream);    
    }

    template void max_gpu<uchar >(const DevMem2D& src1, uchar src2, const DevMem2D& dst, hipStream_t stream);
    template void max_gpu<schar >(const DevMem2D_<schar>& src1, schar src2, const DevMem2D_<schar>& dst, hipStream_t stream);
    template void max_gpu<ushort>(const DevMem2D_<ushort>& src1, ushort src2, const DevMem2D_<ushort>& dst, hipStream_t stream);
    template void max_gpu<short >(const DevMem2D_<short>& src1, short src2, const DevMem2D_<short>& dst, hipStream_t stream);
    template void max_gpu<int   >(const DevMem2D_<int>& src1, int src2, const DevMem2D_<int>& dst, hipStream_t stream);
    template void max_gpu<float >(const DevMem2D_<float>& src1, float src2, const DevMem2D_<float>& dst, hipStream_t stream);
    template void max_gpu<double>(const DevMem2D_<double>& src1, double src2, const DevMem2D_<double>& dst, hipStream_t stream);

    
    //////////////////////////////////////////////////////////////////////////
    // threshold  

    template <template <typename> class Op, typename T>
    void threshold_caller(const DevMem2D_<T>& src, const DevMem2D_<T>& dst, T thresh, T maxVal, 
        hipStream_t stream)
    {
        Op<T> op(thresh, maxVal);
        transform(src, dst, op, stream);
    }

    template <typename T>
    void threshold_gpu(const DevMem2D& src, const DevMem2D& dst, T thresh, T maxVal, int type,
        hipStream_t stream)
    {
        typedef void (*caller_t)(const DevMem2D_<T>& src, const DevMem2D_<T>& dst, T thresh, T maxVal, 
            hipStream_t stream);

        static const caller_t callers[] = 
        {
            threshold_caller<thresh_binary_func, T>, 
            threshold_caller<thresh_binary_inv_func, T>, 
            threshold_caller<thresh_trunc_func, T>, 
            threshold_caller<thresh_to_zero_func, T>, 
            threshold_caller<thresh_to_zero_inv_func, T>
        };

        callers[type]((DevMem2D_<T>)src, (DevMem2D_<T>)dst, thresh, maxVal, stream);
    }

    template void threshold_gpu<uchar>(const DevMem2D& src, const DevMem2D& dst, uchar thresh, uchar maxVal, int type, hipStream_t stream);
    template void threshold_gpu<schar>(const DevMem2D& src, const DevMem2D& dst, schar thresh, schar maxVal, int type, hipStream_t stream);
    template void threshold_gpu<ushort>(const DevMem2D& src, const DevMem2D& dst, ushort thresh, ushort maxVal, int type, hipStream_t stream);
    template void threshold_gpu<short>(const DevMem2D& src, const DevMem2D& dst, short thresh, short maxVal, int type, hipStream_t stream);
    template void threshold_gpu<int>(const DevMem2D& src, const DevMem2D& dst, int thresh, int maxVal, int type, hipStream_t stream);
    template void threshold_gpu<float>(const DevMem2D& src, const DevMem2D& dst, float thresh, float maxVal, int type, hipStream_t stream);
    template void threshold_gpu<double>(const DevMem2D& src, const DevMem2D& dst, double thresh, double maxVal, int type, hipStream_t stream);


    //////////////////////////////////////////////////////////////////////////
    // subtract

    template <typename T>
    void subtractCaller(const DevMem2D src1, const DevMem2D src2, DevMem2D dst, hipStream_t stream)
    {
        transform((DevMem2D_<T>)src1, (DevMem2D_<T>)src2, (DevMem2D_<T>)dst, minus<T>(), stream);
    }

    template void subtractCaller<short>(const DevMem2D src1, const DevMem2D src2, DevMem2D dst, hipStream_t stream);


    //////////////////////////////////////////////////////////////////////////
    // pow
    
    template<typename T, bool Signed = device::numeric_limits<T>::is_signed> struct PowOp : unary_function<T, T>
    {    
        float power;
        PowOp(float power_) : power(power_) {}
            
        __device__ __forceinline__ T operator()(const T& e) const
        {      
            return saturate_cast<T>(__powf((float)e, power));
        }      
    };

    template<typename T> struct PowOp<T, true> : unary_function<T, T>
    {
        float power;
        PowOp(float power_) : power(power_) {}

        __device__ __forceinline__ float operator()(const T& e) const
        {
            T res = saturate_cast<T>(__powf((float)e, power));            
            
            if ( (e < 0) && (1 & (int)power) )
                    res *= -1;            
            return res;         
        }
    };

    template<> struct PowOp<float> : unary_function<float, float>
    {
        float power;
        PowOp(float power_) : power(power_) {}

        __device__ __forceinline__ float operator()(const float& e) const
        {
            return __powf(fabs(e), power);
        }
    };

    template<typename T>
    void pow_caller(const DevMem2D& src, float power, DevMem2D dst, hipStream_t stream)
    {
        transform((DevMem2D_<T>)src, (DevMem2D_<T>)dst, PowOp<T>(power), stream);
    }   

    template void pow_caller<uchar>(const DevMem2D& src, float power, DevMem2D dst, hipStream_t stream);
    template void pow_caller<schar>(const DevMem2D& src, float power, DevMem2D dst, hipStream_t stream);
    template void pow_caller<short>(const DevMem2D& src, float power, DevMem2D dst, hipStream_t stream);
    template void pow_caller<ushort>(const DevMem2D& src, float power, DevMem2D dst, hipStream_t stream);
    template void pow_caller<int>(const DevMem2D& src, float power, DevMem2D dst, hipStream_t stream);
    template void pow_caller<uint>(const DevMem2D& src, float power, DevMem2D dst, hipStream_t stream);
    template void pow_caller<float>(const DevMem2D& src, float power, DevMem2D dst, hipStream_t stream);
}}}
