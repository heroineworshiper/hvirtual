#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/border_interpolate.hpp"

using namespace cv::gpu;
using namespace cv::gpu::device;

/////////////////////////////////// Remap ///////////////////////////////////////////////
namespace cv { namespace gpu { namespace imgproc
{
    texture<unsigned char, 2, hipReadModeNormalizedFloat> tex_remap(0, hipFilterModeLinear, hipAddressModeWrap);

    __global__ void remap_1c(const float* mapx, const float* mapy, size_t map_step, uchar* out, size_t out_step, int width, int height)
    {    
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        int y = blockDim.y * blockIdx.y + threadIdx.y;
        if (x < width && y < height)
        {
            int idx = y * (map_step >> 2) + x; /* map_step >> 2  <=> map_step / sizeof(float)*/

            float xcoo = mapx[idx];
            float ycoo = mapy[idx];

            out[y * out_step + x] = (unsigned char)(255.f * tex2D(tex_remap, xcoo, ycoo));            
        }
    }

    __global__ void remap_3c(const uchar* src, size_t src_step, const float* mapx, const float* mapy,
                             size_t map_step, uchar* dst, size_t dst_step, int width, int height)
    {    
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < width && y < height)
        {
            const int idx = y * (map_step >> 2) + x; /* map_step >> 2  <=> map_step / sizeof(float)*/

            const float xcoo = mapx[idx];
            const float ycoo = mapy[idx];
            
            uchar3 out = make_uchar3(0, 0, 0);

            if (xcoo >= 0 && xcoo < width - 1 && ycoo >= 0 && ycoo < height - 1)
            {
                const int x1 = __float2int_rd(xcoo);
                const int y1 = __float2int_rd(ycoo);
                const int x2 = x1 + 1;
                const int y2 = y1 + 1;
                
                uchar src_reg = *(src + y1 * src_step + 3 * x1);
                out.x += src_reg * (x2 - xcoo) * (y2 - ycoo);
                src_reg = *(src + y1 * src_step + 3 * x1 + 1);
                out.y += src_reg * (x2 - xcoo) * (y2 - ycoo);
                src_reg = *(src + y1 * src_step + 3 * x1 + 2);
                out.z += src_reg * (x2 - xcoo) * (y2 - ycoo);

                src_reg = *(src + y1 * src_step + 3 * x2);                
                out.x += src_reg * (xcoo - x1) * (y2 - ycoo);
                src_reg = *(src + y1 * src_step + 3 * x2 + 1); 
                out.y += src_reg * (xcoo - x1) * (y2 - ycoo);
                src_reg = *(src + y1 * src_step + 3 * x2 + 2); 
                out.z += src_reg * (xcoo - x1) * (y2 - ycoo);

                src_reg = *(src + y2 * src_step + 3 * x1);                
                out.x += src_reg * (x2 - xcoo) * (ycoo - y1);
                src_reg = *(src + y2 * src_step + 3 * x1 + 1); 
                out.y += src_reg * (x2 - xcoo) * (ycoo - y1);
                src_reg = *(src + y2 * src_step + 3 * x1 + 2); 
                out.z += src_reg * (x2 - xcoo) * (ycoo - y1);

                src_reg = *(src + y2 * src_step + 3 * x2);                
                out.x += src_reg * (xcoo - x1) * (ycoo - y1);
                src_reg = *(src + y2 * src_step + 3 * x2 + 1);  
                out.y += src_reg * (xcoo - x1) * (ycoo - y1);
                src_reg = *(src + y2 * src_step + 3 * x2 + 2);  
                out.z += src_reg * (xcoo - x1) * (ycoo - y1);
            }

            /**(uchar3*)(dst + y * dst_step + 3 * x) = out;*/
            *(dst + y * dst_step + 3 * x) = out.x;
            *(dst + y * dst_step + 3 * x + 1) = out.y;
            *(dst + y * dst_step + 3 * x + 2) = out.z;
        }
    }

    void remap_gpu_1c(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, DevMem2D dst)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(dst.cols, threads.x);
        grid.y = divUp(dst.rows, threads.y);

        TextureBinder tex(&tex_remap, src);

        remap_1c<<<grid, threads>>>(xmap.data, ymap.data, xmap.step, dst.data, dst.step, dst.cols, dst.rows);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }
    
    void remap_gpu_3c(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, DevMem2D dst)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(dst.cols, threads.x);
        grid.y = divUp(dst.rows, threads.y);

        remap_3c<<<grid, threads>>>(src.data, src.step, xmap.data, ymap.data, xmap.step, dst.data, dst.step, dst.cols, dst.rows);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }

/////////////////////////////////// MeanShiftfiltering ///////////////////////////////////////////////

    texture<uchar4, 2> tex_meanshift;

    __device__ short2 do_mean_shift(int x0, int y0, unsigned char* out, 
                                    size_t out_step, int cols, int rows, 
                                    int sp, int sr, int maxIter, float eps)
    {
        int isr2 = sr*sr;
        uchar4 c = tex2D(tex_meanshift, x0, y0 );

        // iterate meanshift procedure
        for( int iter = 0; iter < maxIter; iter++ )
        {
            int count = 0;
            int s0 = 0, s1 = 0, s2 = 0, sx = 0, sy = 0;
            float icount;

            //mean shift: process pixels in window (p-sigmaSp)x(p+sigmaSp)
            int minx = x0-sp;
            int miny = y0-sp;
            int maxx = x0+sp;
            int maxy = y0+sp;

            for( int y = miny; y <= maxy; y++)
            {
                int rowCount = 0;
                for( int x = minx; x <= maxx; x++ )
                {                    
                    uchar4 t = tex2D( tex_meanshift, x, y );

                    int norm2 = (t.x - c.x) * (t.x - c.x) + (t.y - c.y) * (t.y - c.y) + (t.z - c.z) * (t.z - c.z);
                    if( norm2 <= isr2 )
                    {
                        s0 += t.x; s1 += t.y; s2 += t.z;
                        sx += x; rowCount++;
                    }
                }
                count += rowCount;
                sy += y*rowCount;
            }

            if( count == 0 )
                break;

            icount = 1.f/count;
            int x1 = __float2int_rz(sx*icount);
            int y1 = __float2int_rz(sy*icount);
            s0 = __float2int_rz(s0*icount);
            s1 = __float2int_rz(s1*icount);
            s2 = __float2int_rz(s2*icount);

            int norm2 = (s0 - c.x) * (s0 - c.x) + (s1 - c.y) * (s1 - c.y) + (s2 - c.z) * (s2 - c.z);

            bool stopFlag = (x0 == x1 && y0 == y1) || (abs(x1-x0) + abs(y1-y0) + norm2 <= eps);

            x0 = x1; y0 = y1;
            c.x = s0; c.y = s1; c.z = s2;

            if( stopFlag )
                break;
        }

        int base = (blockIdx.y * blockDim.y + threadIdx.y) * out_step + (blockIdx.x * blockDim.x + threadIdx.x) * 4 * sizeof(uchar);
        *(uchar4*)(out + base) = c;

        return make_short2((short)x0, (short)y0);
    }

    extern "C" __global__ void meanshift_kernel( unsigned char* out, size_t out_step, int cols, int rows, 
                                                 int sp, int sr, int maxIter, float eps )
    {
        int x0 = blockIdx.x * blockDim.x + threadIdx.x;
        int y0 = blockIdx.y * blockDim.y + threadIdx.y;

        if( x0 < cols && y0 < rows )
            do_mean_shift(x0, y0, out, out_step, cols, rows, sp, sr, maxIter, eps);
    }

    extern "C" __global__ void meanshiftproc_kernel( unsigned char* outr, size_t outrstep, 
                                                 unsigned char* outsp, size_t outspstep, 
                                                 int cols, int rows, 
                                                 int sp, int sr, int maxIter, float eps )
    {
        int x0 = blockIdx.x * blockDim.x + threadIdx.x;
        int y0 = blockIdx.y * blockDim.y + threadIdx.y;

        if( x0 < cols && y0 < rows )
        {            
            int basesp = (blockIdx.y * blockDim.y + threadIdx.y) * outspstep + (blockIdx.x * blockDim.x + threadIdx.x) * 2 * sizeof(short);
            *(short2*)(outsp + basesp) = do_mean_shift(x0, y0, outr, outrstep, cols, rows, sp, sr, maxIter, eps);
        }
    }

    extern "C" void meanShiftFiltering_gpu(const DevMem2D& src, DevMem2D dst, int sp, int sr, int maxIter, float eps)
    {
        dim3 grid(1, 1, 1);
        dim3 threads(32, 8, 1);
        grid.x = divUp(src.cols, threads.x);
        grid.y = divUp(src.rows, threads.y);

        hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
        cudaSafeCall( hipBindTexture2D( 0, tex_meanshift, src.data, desc, src.cols, src.rows, src.step ) );

        meanshift_kernel<<< grid, threads >>>( dst.data, dst.step, dst.cols, dst.rows, sp, sr, maxIter, eps );
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
        cudaSafeCall( hipUnbindTexture( tex_meanshift ) );        
    }
    extern "C" void meanShiftProc_gpu(const DevMem2D& src, DevMem2D dstr, DevMem2D dstsp, int sp, int sr, int maxIter, float eps) 
    {
        dim3 grid(1, 1, 1);
        dim3 threads(32, 8, 1);
        grid.x = divUp(src.cols, threads.x);
        grid.y = divUp(src.rows, threads.y);

        hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
        cudaSafeCall( hipBindTexture2D( 0, tex_meanshift, src.data, desc, src.cols, src.rows, src.step ) );

        meanshiftproc_kernel<<< grid, threads >>>( dstr.data, dstr.step, dstsp.data, dstsp.step, dstr.cols, dstr.rows, sp, sr, maxIter, eps );
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
        cudaSafeCall( hipUnbindTexture( tex_meanshift ) );        
    }

/////////////////////////////////// drawColorDisp ///////////////////////////////////////////////

    template <typename T>
    __device__ unsigned int cvtPixel(T d, int ndisp, float S = 1, float V = 1)
    {        
        unsigned int H = ((ndisp-d) * 240)/ndisp;

        unsigned int hi = (H/60) % 6;
        float f = H/60.f - H/60;
        float p = V * (1 - S);
        float q = V * (1 - f * S);
        float t = V * (1 - (1 - f) * S);

        float3 res;
        
        if (hi == 0) //R = V,	G = t,	B = p
        {
            res.x = p;
            res.y = t;
            res.z = V;
        }

        if (hi == 1) // R = q,	G = V,	B = p
        {
            res.x = p;
            res.y = V;
            res.z = q;
        }        
        
        if (hi == 2) // R = p,	G = V,	B = t
        {
            res.x = t;
            res.y = V;
            res.z = p;
        }
            
        if (hi == 3) // R = p,	G = q,	B = V
        {
            res.x = V;
            res.y = q;
            res.z = p;
        }

        if (hi == 4) // R = t,	G = p,	B = V
        {
            res.x = V;
            res.y = p;
            res.z = t;
        }

        if (hi == 5) // R = V,	G = p,	B = q
        {
            res.x = q;
            res.y = p;
            res.z = V;
        }
        const unsigned int b = (unsigned int)(max(0.f, min (res.x, 1.f)) * 255.f);
        const unsigned int g = (unsigned int)(max(0.f, min (res.y, 1.f)) * 255.f);
        const unsigned int r = (unsigned int)(max(0.f, min (res.z, 1.f)) * 255.f);
        const unsigned int a = 255U;

        return (a << 24) + (r << 16) + (g << 8) + b;    
    } 

    __global__ void drawColorDisp(uchar* disp, size_t disp_step, uchar* out_image, size_t out_step, int width, int height, int ndisp)
    {
        const int x = (blockIdx.x * blockDim.x + threadIdx.x) << 2;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < width && y < height) 
        {
            uchar4 d4 = *(uchar4*)(disp + y * disp_step + x);

            uint4 res;
            res.x = cvtPixel(d4.x, ndisp);
            res.y = cvtPixel(d4.y, ndisp);
            res.z = cvtPixel(d4.z, ndisp);
            res.w = cvtPixel(d4.w, ndisp);
                    
            uint4* line = (uint4*)(out_image + y * out_step);
            line[x >> 2] = res;
        }
    }

    __global__ void drawColorDisp(short* disp, size_t disp_step, uchar* out_image, size_t out_step, int width, int height, int ndisp)
    {
        const int x = (blockIdx.x * blockDim.x + threadIdx.x) << 1;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < width && y < height) 
        {
            short2 d2 = *(short2*)(disp + y * disp_step + x);

            uint2 res;
            res.x = cvtPixel(d2.x, ndisp);            
            res.y = cvtPixel(d2.y, ndisp);

            uint2* line = (uint2*)(out_image + y * out_step);
            line[x >> 1] = res;
        }
    }


    void drawColorDisp_gpu(const DevMem2D& src, const DevMem2D& dst, int ndisp, const hipStream_t& stream)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(src.cols, threads.x << 2);
        grid.y = divUp(src.rows, threads.y);
         
        drawColorDisp<<<grid, threads, 0, stream>>>(src.data, src.step, dst.data, dst.step, src.cols, src.rows, ndisp);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() ); 
    }

    void drawColorDisp_gpu(const DevMem2D_<short>& src, const DevMem2D& dst, int ndisp, const hipStream_t& stream)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(src.cols, threads.x << 1);
        grid.y = divUp(src.rows, threads.y);
         
        drawColorDisp<<<grid, threads, 0, stream>>>(src.data, src.step / sizeof(short), dst.data, dst.step, src.cols, src.rows, ndisp);
        cudaSafeCall( hipGetLastError() );
        
        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

/////////////////////////////////// reprojectImageTo3D ///////////////////////////////////////////////

    __constant__ float cq[16];

    template <typename T>
    __global__ void reprojectImageTo3D(const T* disp, size_t disp_step, float* xyzw, size_t xyzw_step, int rows, int cols)
    {        
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (y < rows && x < cols)
        {

            float qx = cq[1] * y + cq[3], qy = cq[5] * y + cq[7];
            float qz = cq[9] * y + cq[11], qw = cq[13] * y + cq[15];

            qx += x * cq[0]; 
            qy += x * cq[4];
            qz += x * cq[8];
            qw += x * cq[12];

            T d = *(disp + disp_step * y + x);

            float iW = 1.f / (qw + cq[14] * d);
            float4 v;
            v.x = (qx + cq[2] * d) * iW;
            v.y = (qy + cq[6] * d) * iW;
            v.z = (qz + cq[10] * d) * iW;
            v.w = 1.f;

            *(float4*)(xyzw + xyzw_step * y + (x * 4)) = v;
        }
    }

    template <typename T>
    inline void reprojectImageTo3D_caller(const DevMem2D_<T>& disp, const DevMem2Df& xyzw, const float* q, const hipStream_t& stream)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(disp.cols, threads.x);
        grid.y = divUp(disp.rows, threads.y);

        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cq), q, 16 * sizeof(float)) );

        reprojectImageTo3D<<<grid, threads, 0, stream>>>(disp.data, disp.step / sizeof(T), xyzw.data, xyzw.step / sizeof(float), disp.rows, disp.cols);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void reprojectImageTo3D_gpu(const DevMem2D& disp, const DevMem2Df& xyzw, const float* q, const hipStream_t& stream)
    {
        reprojectImageTo3D_caller(disp, xyzw, q, stream);
    }

    void reprojectImageTo3D_gpu(const DevMem2D_<short>& disp, const DevMem2Df& xyzw, const float* q, const hipStream_t& stream)
    {
        reprojectImageTo3D_caller(disp, xyzw, q, stream);
    }

//////////////////////////////////////// Extract Cov Data ////////////////////////////////////////////////

    __global__ void extractCovData_kernel(const int cols, const int rows, const PtrStepf Dx, 
                                          const PtrStepf Dy, PtrStepf dst)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < cols && y < rows)
        {            
            float dx = Dx.ptr(y)[x];
            float dy = Dy.ptr(y)[x];

            dst.ptr(y)[x] = dx * dx;
            dst.ptr(y + rows)[x] = dx * dy;
            dst.ptr(y + (rows << 1))[x] = dy * dy;
        }
    }

    void extractCovData_caller(const DevMem2Df Dx, const DevMem2Df Dy, PtrStepf dst)
    {
        dim3 threads(32, 8);
        dim3 grid(divUp(Dx.cols, threads.x), divUp(Dx.rows, threads.y));

        extractCovData_kernel<<<grid, threads>>>(Dx.cols, Dx.rows, Dx, Dy, dst);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }

/////////////////////////////////////////// Corner Harris /////////////////////////////////////////////////

    texture<float, 2> harrisDxTex;
    texture<float, 2> harrisDyTex;

    __global__ void cornerHarris_kernel(const int cols, const int rows, const int block_size, const float k,
                                        PtrStep dst)
    {
        const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < cols && y < rows)
        {
            float a = 0.f;
            float b = 0.f;
            float c = 0.f;

            const int ibegin = y - (block_size / 2);
            const int jbegin = x - (block_size / 2);
            const int iend = ibegin + block_size;
            const int jend = jbegin + block_size;

            for (int i = ibegin; i < iend; ++i)
            {
                for (int j = jbegin; j < jend; ++j)
                {
                    float dx = tex2D(harrisDxTex, j, i);
                    float dy = tex2D(harrisDyTex, j, i);
                    a += dx * dx;
                    b += dx * dy;
                    c += dy * dy;
                }
            }

            ((float*)dst.ptr(y))[x] = a * c - b * b - k * (a + c) * (a + c);
        }
    }

    template <typename B>
    __global__ void cornerHarris_kernel(const int cols, const int rows, const int block_size, const float k,
                                        PtrStep dst, B border_row, B border_col)
    {
        const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < cols && y < rows)
        {
            float a = 0.f;
            float b = 0.f;
            float c = 0.f;

            const int ibegin = y - (block_size / 2);
            const int jbegin = x - (block_size / 2);
            const int iend = ibegin + block_size;
            const int jend = jbegin + block_size;

            for (int i = ibegin; i < iend; ++i)
            {
                int y = border_col.idx(i);
                for (int j = jbegin; j < jend; ++j)
                {
                    int x = border_row.idx(j);
                    float dx = tex2D(harrisDxTex, x, y);
                    float dy = tex2D(harrisDyTex, x, y);
                    a += dx * dx;
                    b += dx * dy;
                    c += dy * dy;
                }
            }

            ((float*)dst.ptr(y))[x] = a * c - b * b - k * (a + c) * (a + c);
        }
    }

    void cornerHarris_caller(const int block_size, const float k, const DevMem2D Dx, const DevMem2D Dy, DevMem2D dst, 
                             int border_type)
    {
        const int rows = Dx.rows;
        const int cols = Dx.cols;

        dim3 threads(32, 8);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
        hipBindTexture2D(0, harrisDxTex, Dx.data, desc, Dx.cols, Dx.rows, Dx.step);
        hipBindTexture2D(0, harrisDyTex, Dy.data, desc, Dy.cols, Dy.rows, Dy.step);
        harrisDxTex.filterMode = hipFilterModePoint;
        harrisDyTex.filterMode = hipFilterModePoint;

        switch (border_type) 
        {
        case BORDER_REFLECT101_GPU:
            cornerHarris_kernel<<<grid, threads>>>(
                    cols, rows, block_size, k, dst, BrdReflect101(cols), BrdReflect101(rows));
            break;
        case BORDER_REPLICATE_GPU:
            harrisDxTex.addressMode[0] = hipAddressModeClamp;
            harrisDxTex.addressMode[1] = hipAddressModeClamp;
            harrisDyTex.addressMode[0] = hipAddressModeClamp;
            harrisDyTex.addressMode[1] = hipAddressModeClamp;
            cornerHarris_kernel<<<grid, threads>>>(cols, rows, block_size, k, dst);
            break;
        }

        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );

        cudaSafeCall(hipUnbindTexture(harrisDxTex));
        cudaSafeCall(hipUnbindTexture(harrisDyTex));
    }

/////////////////////////////////////////// Corner Min Eigen Val /////////////////////////////////////////////////

    texture<float, 2> minEigenValDxTex;
    texture<float, 2> minEigenValDyTex;

    __global__ void cornerMinEigenVal_kernel(const int cols, const int rows, const int block_size, 
                                             PtrStep dst)
    {
        const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < cols && y < rows)
        {
            float a = 0.f;
            float b = 0.f;
            float c = 0.f;

            const int ibegin = y - (block_size / 2);
            const int jbegin = x - (block_size / 2);
            const int iend = ibegin + block_size;
            const int jend = jbegin + block_size;

            for (int i = ibegin; i < iend; ++i)
            {
                for (int j = jbegin; j < jend; ++j)
                {
                    float dx = tex2D(minEigenValDxTex, j, i);
                    float dy = tex2D(minEigenValDyTex, j, i);
                    a += dx * dx;
                    b += dx * dy;
                    c += dy * dy;
                }
            }

            a *= 0.5f;
            c *= 0.5f;
            ((float*)dst.ptr(y))[x] = (a + c) - sqrtf((a - c) * (a - c) + b * b);
        }
    }


    template <typename B>
    __global__ void cornerMinEigenVal_kernel(const int cols, const int rows, const int block_size, 
                                             PtrStep dst, B border_row, B border_col)
    {
        const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < cols && y < rows)
        {
            float a = 0.f;
            float b = 0.f;
            float c = 0.f;

            const int ibegin = y - (block_size / 2);
            const int jbegin = x - (block_size / 2);
            const int iend = ibegin + block_size;
            const int jend = jbegin + block_size;

            for (int i = ibegin; i < iend; ++i)
            {
                int y = border_col.idx(i);
                for (int j = jbegin; j < jend; ++j)
                {
                    int x = border_row.idx(j);
                    float dx = tex2D(minEigenValDxTex, x, y);
                    float dy = tex2D(minEigenValDyTex, x, y);
                    a += dx * dx;
                    b += dx * dy;
                    c += dy * dy;
                }
            }

            a *= 0.5f;
            c *= 0.5f;
            ((float*)dst.ptr(y))[x] = (a + c) - sqrtf((a - c) * (a - c) + b * b);
        }
    }

    void cornerMinEigenVal_caller(const int block_size, const DevMem2D Dx, const DevMem2D Dy, DevMem2D dst,
                                  int border_type)
    {
        const int rows = Dx.rows;
        const int cols = Dx.cols;

        dim3 threads(32, 8);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
        hipBindTexture2D(0, minEigenValDxTex, Dx.data, desc, Dx.cols, Dx.rows, Dx.step);
        hipBindTexture2D(0, minEigenValDyTex, Dy.data, desc, Dy.cols, Dy.rows, Dy.step);
        minEigenValDxTex.filterMode = hipFilterModePoint;
        minEigenValDyTex.filterMode = hipFilterModePoint;

        switch (border_type)
        {
        case BORDER_REFLECT101_GPU:
            cornerMinEigenVal_kernel<<<grid, threads>>>(
                    cols, rows, block_size, dst, BrdReflect101(cols), BrdReflect101(rows));
            break;
        case BORDER_REPLICATE_GPU:
            minEigenValDxTex.addressMode[0] = hipAddressModeClamp;
            minEigenValDxTex.addressMode[1] = hipAddressModeClamp;
            minEigenValDyTex.addressMode[0] = hipAddressModeClamp;
            minEigenValDyTex.addressMode[1] = hipAddressModeClamp;
            cornerMinEigenVal_kernel<<<grid, threads>>>(cols, rows, block_size, dst);
            break;
        }

        cudaSafeCall( hipGetLastError() );

        cudaSafeCall(hipDeviceSynchronize());

        cudaSafeCall(hipUnbindTexture(minEigenValDxTex));
        cudaSafeCall(hipUnbindTexture(minEigenValDyTex));
    }

////////////////////////////// Column Sum //////////////////////////////////////

    __global__ void column_sumKernel_32F(int cols, int rows, const PtrStep src, const PtrStep dst)
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;

        if (x < cols)
        {
            const unsigned char* src_data = src.data + x * sizeof(float);
            unsigned char* dst_data = dst.data + x * sizeof(float);

            float sum = 0.f;
            for (int y = 0; y < rows; ++y)
            {
                sum += *(const float*)src_data;
                *(float*)dst_data = sum;
                src_data += src.step;
                dst_data += dst.step;
            }
        }
    }


    void columnSum_32F(const DevMem2D src, const DevMem2D dst)
    {
        dim3 threads(256);
        dim3 grid(divUp(src.cols, threads.x));

        column_sumKernel_32F<<<grid, threads>>>(src.cols, src.rows, src, dst);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }


    //////////////////////////////////////////////////////////////////////////
    // mulSpectrums

    __global__ void mulSpectrumsKernel(const PtrStep_<hipfftComplex> a, const PtrStep_<hipfftComplex> b, 
                                       DevMem2D_<hipfftComplex> c)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;    
        const int y = blockIdx.y * blockDim.y + threadIdx.y;    

        if (x < c.cols && y < c.rows) 
        {
            c.ptr(y)[x] = hipCmulf(a.ptr(y)[x], b.ptr(y)[x]);
        }
    }


    void mulSpectrums(const PtrStep_<hipfftComplex> a, const PtrStep_<hipfftComplex> b, 
                      DevMem2D_<hipfftComplex> c)
    {
        dim3 threads(256);
        dim3 grid(divUp(c.cols, threads.x), divUp(c.rows, threads.y));

        mulSpectrumsKernel<<<grid, threads>>>(a, b, c);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }


    //////////////////////////////////////////////////////////////////////////
    // mulSpectrums_CONJ

    __global__ void mulSpectrumsKernel_CONJ(
            const PtrStep_<hipfftComplex> a, const PtrStep_<hipfftComplex> b,
            DevMem2D_<hipfftComplex> c)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;    
        const int y = blockIdx.y * blockDim.y + threadIdx.y;    

        if (x < c.cols && y < c.rows) 
        {
            c.ptr(y)[x] = hipCmulf(a.ptr(y)[x], hipConjf(b.ptr(y)[x]));
        }
    }


    void mulSpectrums_CONJ(const PtrStep_<hipfftComplex> a, const PtrStep_<hipfftComplex> b, 
                           DevMem2D_<hipfftComplex> c)
    {
        dim3 threads(256);
        dim3 grid(divUp(c.cols, threads.x), divUp(c.rows, threads.y));

        mulSpectrumsKernel_CONJ<<<grid, threads>>>(a, b, c);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }


    //////////////////////////////////////////////////////////////////////////
    // mulAndScaleSpectrums

    __global__ void mulAndScaleSpectrumsKernel(
            const PtrStep_<hipfftComplex> a, const PtrStep_<hipfftComplex> b, 
            float scale, DevMem2D_<hipfftComplex> c)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < c.cols && y < c.rows) 
        {
            hipfftComplex v = hipCmulf(a.ptr(y)[x], b.ptr(y)[x]);
            c.ptr(y)[x] = make_hipFloatComplex(hipCrealf(v) * scale, hipCimagf(v) * scale);
        }
    }


    void mulAndScaleSpectrums(const PtrStep_<hipfftComplex> a, const PtrStep_<hipfftComplex> b,
                              float scale, DevMem2D_<hipfftComplex> c)
    {
        dim3 threads(256);
        dim3 grid(divUp(c.cols, threads.x), divUp(c.rows, threads.y));

        mulAndScaleSpectrumsKernel<<<grid, threads>>>(a, b, scale, c);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }


    //////////////////////////////////////////////////////////////////////////
    // mulAndScaleSpectrums_CONJ

    __global__ void mulAndScaleSpectrumsKernel_CONJ(
            const PtrStep_<hipfftComplex> a, const PtrStep_<hipfftComplex> b,
            float scale, DevMem2D_<hipfftComplex> c)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < c.cols && y < c.rows) 
        {
            hipfftComplex v = hipCmulf(a.ptr(y)[x], hipConjf(b.ptr(y)[x]));
            c.ptr(y)[x] = make_hipFloatComplex(hipCrealf(v) * scale, hipCimagf(v) * scale);
        }
    }


    void mulAndScaleSpectrums_CONJ(const PtrStep_<hipfftComplex> a, const PtrStep_<hipfftComplex> b,
                                  float scale, DevMem2D_<hipfftComplex> c)
    {
        dim3 threads(256);
        dim3 grid(divUp(c.cols, threads.x), divUp(c.rows, threads.y));

        mulAndScaleSpectrumsKernel_CONJ<<<grid, threads>>>(a, b, scale, c);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }


    /////////////////////////////////////////////////////////////////////////
    // downsample

    template <typename T, int cn>
    __global__ void downsampleKernel(const PtrStep_<T> src, DevMem2D_<T> dst)
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < dst.cols && y < dst.rows)
        {
            int ch_x = x / cn;
            dst.ptr(y)[x] = src.ptr(y*2)[ch_x*2*cn + x - ch_x*cn];
        }
    }


    template <typename T, int cn>
    void downsampleCaller(const DevMem2D src, DevMem2D dst, hipStream_t stream)
    {
        dim3 threads(32, 8);
        dim3 grid(divUp(dst.cols, threads.x), divUp(dst.rows, threads.y));

        downsampleKernel<T,cn><<<grid, threads, 0, stream>>>(DevMem2D_<T>(src), DevMem2D_<T>(dst));
        cudaSafeCall(hipGetLastError());
        
        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


    template void downsampleCaller<uchar,1>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<uchar,2>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<uchar,3>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<uchar,4>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<short,1>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<short,2>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<short,3>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<short,4>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<float,1>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<float,2>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<float,3>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<float,4>(const DevMem2D src, DevMem2D dst, hipStream_t stream);


    //////////////////////////////////////////////////////////////////////////
    // upsample

    template <typename T, int cn>
    __global__ void upsampleKernel(const PtrStep_<T> src, DevMem2D_<T> dst)
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < dst.cols && y < dst.rows)
        {
            int ch_x = x / cn;
            T val = ((ch_x & 1) || (y & 1)) ? 0 : src.ptr(y/2)[ch_x/2*cn + x - ch_x*cn];
            dst.ptr(y)[x] = val;
        }
    }


    template <typename T, int cn>
    void upsampleCaller(const DevMem2D src, DevMem2D dst, hipStream_t stream)
    {
        dim3 threads(32, 8);
        dim3 grid(divUp(dst.cols, threads.x), divUp(dst.rows, threads.y));

        upsampleKernel<T,cn><<<grid, threads, 0, stream>>>(DevMem2D_<T>(src), DevMem2D_<T>(dst));
        cudaSafeCall(hipGetLastError());

        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


    template void upsampleCaller<uchar,1>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<uchar,2>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<uchar,3>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<uchar,4>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<short,1>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<short,2>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<short,3>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<short,4>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<float,1>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<float,2>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<float,3>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<float,4>(const DevMem2D src, DevMem2D dst, hipStream_t stream);


    //////////////////////////////////////////////////////////////////////////
    // buildWarpMaps

    namespace build_warp_maps
    {
        __constant__ float cr[9];
        __constant__ float crinv[9];
        __constant__ float cf, cs;
        __constant__ float chalf_w, chalf_h;
        __constant__ float cdist;
    }


    class PlaneMapper
    {
    public:
        static __device__ __forceinline__ void mapBackward(float u, float v, float &x, float &y)
        {
            using namespace build_warp_maps;

            float x_ = u / cs;
            float y_ = v / cs;

            float z;
            x = crinv[0]*x_ + crinv[1]*y_ + crinv[2]*cdist;
            y = crinv[3]*x_ + crinv[4]*y_ + crinv[5]*cdist;
            z = crinv[6]*x_ + crinv[7]*y_ + crinv[8]*cdist;

            x = cf*x/z + chalf_w;
            y = cf*y/z + chalf_h;
        }
    };


    class CylindricalMapper
    {
    public:
        static __device__ __forceinline__ void mapBackward(float u, float v, float &x, float &y)
        {
            using namespace build_warp_maps;

            u /= cs;
            float x_ = sinf(u);
            float y_ = v / cs;
            float z_ = cosf(u);

            float z;
            x = crinv[0]*x_ + crinv[1]*y_ + crinv[2]*z_;
            y = crinv[3]*x_ + crinv[4]*y_ + crinv[5]*z_;
            z = crinv[6]*x_ + crinv[7]*y_ + crinv[8]*z_;

            x = cf*x/z + chalf_w;
            y = cf*y/z + chalf_h;
        }
    };


    class SphericalMapper
    {
    public:
        static __device__ __forceinline__ void mapBackward(float u, float v, float &x, float &y)
        {
            using namespace build_warp_maps;

            v /= cs;
            u /= cs;

            float sinv = sinf(v);
            float x_ = sinv * sinf(u);
            float y_ = -cosf(v);
            float z_ = sinv * cosf(u);

            float z;
            x = crinv[0]*x_ + crinv[1]*y_ + crinv[2]*z_;
            y = crinv[3]*x_ + crinv[4]*y_ + crinv[5]*z_;
            z = crinv[6]*x_ + crinv[7]*y_ + crinv[8]*z_;

            x = cf*x/z + chalf_w;
            y = cf*y/z + chalf_h;
        }
    };


    template <typename Mapper>
    __global__ void buildWarpMapsKernel(int tl_u, int tl_v, int cols, int rows,
                                        PtrStepf map_x, PtrStepf map_y)
    {
        int du = blockIdx.x * blockDim.x + threadIdx.x;
        int dv = blockIdx.y * blockDim.y + threadIdx.y;
        if (du < cols && dv < rows)
        {
            float u = tl_u + du;
            float v = tl_v + dv;
            float x, y;
            Mapper::mapBackward(u, v, x, y);
            map_x.ptr(dv)[du] = x;
            map_y.ptr(dv)[du] = y;
        }
    }


    void buildWarpPlaneMaps(int tl_u, int tl_v, DevMem2Df map_x, DevMem2Df map_y,
                            const float r[9], const float rinv[9], float f, float s, float dist,
                            float half_w, float half_h, hipStream_t stream)
    {
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cr), r, 9*sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::crinv), rinv, 9*sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cf), &f, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cs), &s, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::chalf_w), &half_w, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::chalf_h), &half_h, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cdist), &dist, sizeof(float)));

        int cols = map_x.cols;
        int rows = map_x.rows;

        dim3 threads(32, 8);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        buildWarpMapsKernel<PlaneMapper><<<grid,threads>>>(tl_u, tl_v, cols, rows, map_x, map_y);
        cudaSafeCall(hipGetLastError());
        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


    void buildWarpCylindricalMaps(int tl_u, int tl_v, DevMem2Df map_x, DevMem2Df map_y,
                                  const float r[9], const float rinv[9], float f, float s,
                                  float half_w, float half_h, hipStream_t stream)
    {
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cr), r, 9*sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::crinv), rinv, 9*sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cf), &f, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cs), &s, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::chalf_w), &half_w, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::chalf_h), &half_h, sizeof(float)));

        int cols = map_x.cols;
        int rows = map_x.rows;

        dim3 threads(32, 8);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        buildWarpMapsKernel<CylindricalMapper><<<grid,threads>>>(tl_u, tl_v, cols, rows, map_x, map_y);
        cudaSafeCall(hipGetLastError());
        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


    void buildWarpSphericalMaps(int tl_u, int tl_v, DevMem2Df map_x, DevMem2Df map_y,
                                const float r[9], const float rinv[9], float f, float s,
                                float half_w, float half_h, hipStream_t stream)
    {
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cr), r, 9*sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::crinv), rinv, 9*sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cf), &f, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cs), &s, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::chalf_w), &half_w, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::chalf_h), &half_h, sizeof(float)));

        int cols = map_x.cols;
        int rows = map_x.rows;

        dim3 threads(32, 8);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        buildWarpMapsKernel<SphericalMapper><<<grid,threads>>>(tl_u, tl_v, cols, rows, map_x, map_y);
        cudaSafeCall(hipGetLastError());
        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


}}}


