#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"
#include "opencv2/gpu/device/transform.hpp"
#include "opencv2/gpu/device/functional.hpp"

using namespace cv::gpu::device;

namespace cv { namespace gpu { namespace matrix_operations {

    template <typename T> struct shift_and_sizeof;
    template <> struct shift_and_sizeof<signed char> { enum { shift = 0 }; };
    template <> struct shift_and_sizeof<unsigned char> { enum { shift = 0 }; };
    template <> struct shift_and_sizeof<short> { enum { shift = 1 }; };
    template <> struct shift_and_sizeof<unsigned short> { enum { shift = 1 }; };
    template <> struct shift_and_sizeof<int> { enum { shift = 2 }; };
    template <> struct shift_and_sizeof<float> { enum { shift = 2 }; };
    template <> struct shift_and_sizeof<double> { enum { shift = 3 }; };

///////////////////////////////////////////////////////////////////////////
////////////////////////////////// CopyTo /////////////////////////////////
///////////////////////////////////////////////////////////////////////////

    template<typename T>
    __global__ void copy_to_with_mask(const T* mat_src, T* mat_dst, const uchar* mask, int cols, int rows, size_t step_mat, size_t step_mask, int channels)
    {
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        size_t y = blockIdx.y * blockDim.y + threadIdx.y;

        if ((x < cols * channels ) && (y < rows))
            if (mask[y * step_mask + x / channels] != 0)
            {
                size_t idx = y * ( step_mat >> shift_and_sizeof<T>::shift ) + x;
                mat_dst[idx] = mat_src[idx];
            }
    }
    typedef void (*CopyToFunc)(const DevMem2D& mat_src, const DevMem2D& mat_dst, const DevMem2D& mask, int channels, const hipStream_t & stream);

    template<typename T>
    void copy_to_with_mask_run(const DevMem2D& mat_src, const DevMem2D& mat_dst, const DevMem2D& mask, int channels, const hipStream_t & stream)
    {
        dim3 threadsPerBlock(16,16, 1);
        dim3 numBlocks ( divUp(mat_src.cols * channels , threadsPerBlock.x) , divUp(mat_src.rows , threadsPerBlock.y), 1);

        copy_to_with_mask<T><<<numBlocks,threadsPerBlock, 0, stream>>>
                ((T*)mat_src.data, (T*)mat_dst.data, (unsigned char*)mask.data, mat_src.cols, mat_src.rows, mat_src.step, mask.step, channels);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall ( hipDeviceSynchronize() );
    }

    void copy_to_with_mask(const DevMem2D& mat_src, DevMem2D mat_dst, int depth, const DevMem2D& mask, int channels, const hipStream_t & stream)
    {
        static CopyToFunc tab[8] =
        {
            copy_to_with_mask_run<unsigned char>,
            copy_to_with_mask_run<signed char>,
            copy_to_with_mask_run<unsigned short>,
            copy_to_with_mask_run<short>,
            copy_to_with_mask_run<int>,
            copy_to_with_mask_run<float>,
            copy_to_with_mask_run<double>,
            0
        };

        CopyToFunc func = tab[depth];

        if (func == 0) cv::gpu::error("Unsupported copyTo operation", __FILE__, __LINE__);

        func(mat_src, mat_dst, mask, channels, stream);
    }

///////////////////////////////////////////////////////////////////////////
////////////////////////////////// SetTo //////////////////////////////////
///////////////////////////////////////////////////////////////////////////

    __constant__ uchar scalar_8u[4];
    __constant__ schar scalar_8s[4];
    __constant__ ushort scalar_16u[4];
    __constant__ short scalar_16s[4];
    __constant__ int scalar_32s[4];
    __constant__ float scalar_32f[4]; 
    __constant__ double scalar_64f[4];

    template <typename T> __device__ __forceinline__ T readScalar(int i);
    template <> __device__ __forceinline__ uchar readScalar<uchar>(int i) {return scalar_8u[i];}
    template <> __device__ __forceinline__ schar readScalar<schar>(int i) {return scalar_8s[i];}
    template <> __device__ __forceinline__ ushort readScalar<ushort>(int i) {return scalar_16u[i];}
    template <> __device__ __forceinline__ short readScalar<short>(int i) {return scalar_16s[i];}
    template <> __device__ __forceinline__ int readScalar<int>(int i) {return scalar_32s[i];}
    template <> __device__ __forceinline__ float readScalar<float>(int i) {return scalar_32f[i];}
    template <> __device__ __forceinline__ double readScalar<double>(int i) {return scalar_64f[i];}

    void writeScalar(const uchar* vals)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_8u), vals, sizeof(uchar) * 4) );
    }
    void writeScalar(const schar* vals)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_8s), vals, sizeof(schar) * 4) );
    }
    void writeScalar(const ushort* vals)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_16u), vals, sizeof(ushort) * 4) );
    }
    void writeScalar(const short* vals)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_16s), vals, sizeof(short) * 4) );
    }
    void writeScalar(const int* vals)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_32s), vals, sizeof(int) * 4) );
    }
    void writeScalar(const float* vals)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_32f), vals, sizeof(float) * 4) );
    }
    void writeScalar(const double* vals)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_64f), vals, sizeof(double) * 4) );
    }

    template<typename T>
    __global__ void set_to_without_mask(T* mat, int cols, int rows, size_t step, int channels)
    {
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        size_t y = blockIdx.y * blockDim.y + threadIdx.y;

        if ((x < cols * channels ) && (y < rows))
        {
            size_t idx = y * ( step >> shift_and_sizeof<T>::shift ) + x;
            mat[idx] = readScalar<T>(x % channels);
        }
    }

    template<typename T>
    __global__ void set_to_with_mask(T* mat, const uchar* mask, int cols, int rows, size_t step, int channels, size_t step_mask)
    {
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        size_t y = blockIdx.y * blockDim.y + threadIdx.y;

        if ((x < cols * channels ) && (y < rows))
            if (mask[y * step_mask + x / channels] != 0)
            {
                size_t idx = y * ( step >> shift_and_sizeof<T>::shift ) + x;
                mat[idx] = readScalar<T>(x % channels);
            }
    }
    template <typename T>
    void set_to_gpu(const DevMem2D& mat, const T* scalar, const DevMem2D& mask, int channels, hipStream_t stream)
    {
        writeScalar(scalar);

        dim3 threadsPerBlock(32, 8, 1);
        dim3 numBlocks (mat.cols * channels / threadsPerBlock.x + 1, mat.rows / threadsPerBlock.y + 1, 1);

        set_to_with_mask<T><<<numBlocks, threadsPerBlock, 0, stream>>>((T*)mat.data, (uchar*)mask.data, mat.cols, mat.rows, mat.step, channels, mask.step);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall ( hipDeviceSynchronize() );
    }

    template void set_to_gpu<uchar >(const DevMem2D& mat, const uchar* scalar, const DevMem2D& mask, int channels, hipStream_t stream);
    template void set_to_gpu<schar >(const DevMem2D& mat, const schar* scalar, const DevMem2D& mask, int channels, hipStream_t stream);
    template void set_to_gpu<ushort>(const DevMem2D& mat, const ushort* scalar, const DevMem2D& mask, int channels, hipStream_t stream);
    template void set_to_gpu<short >(const DevMem2D& mat, const short* scalar, const DevMem2D& mask, int channels, hipStream_t stream);
    template void set_to_gpu<int   >(const DevMem2D& mat, const int* scalar, const DevMem2D& mask, int channels, hipStream_t stream);
    template void set_to_gpu<float >(const DevMem2D& mat, const float* scalar, const DevMem2D& mask, int channels, hipStream_t stream);
    template void set_to_gpu<double>(const DevMem2D& mat, const double* scalar, const DevMem2D& mask, int channels, hipStream_t stream);

    template <typename T>
    void set_to_gpu(const DevMem2D& mat, const T* scalar, int channels, hipStream_t stream)
    {
        writeScalar(scalar);

        dim3 threadsPerBlock(32, 8, 1);
        dim3 numBlocks (mat.cols * channels / threadsPerBlock.x + 1, mat.rows / threadsPerBlock.y + 1, 1);

        set_to_without_mask<T><<<numBlocks, threadsPerBlock, 0, stream>>>((T*)mat.data, mat.cols, mat.rows, mat.step, channels);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall ( hipDeviceSynchronize() );
    }

    template void set_to_gpu<uchar >(const DevMem2D& mat, const uchar* scalar, int channels, hipStream_t stream);
    template void set_to_gpu<schar >(const DevMem2D& mat, const schar* scalar, int channels, hipStream_t stream);
    template void set_to_gpu<ushort>(const DevMem2D& mat, const ushort* scalar, int channels, hipStream_t stream);
    template void set_to_gpu<short >(const DevMem2D& mat, const short* scalar, int channels, hipStream_t stream);
    template void set_to_gpu<int   >(const DevMem2D& mat, const int* scalar, int channels, hipStream_t stream);
    template void set_to_gpu<float >(const DevMem2D& mat, const float* scalar, int channels, hipStream_t stream);
    template void set_to_gpu<double>(const DevMem2D& mat, const double* scalar, int channels, hipStream_t stream);

///////////////////////////////////////////////////////////////////////////
//////////////////////////////// ConvertTo ////////////////////////////////
///////////////////////////////////////////////////////////////////////////

    template <typename T, typename D> struct Convertor : unary_function<T, D>
    {
        Convertor(double alpha_, double beta_) : alpha(alpha_), beta(beta_) {}

        __device__ __forceinline__ D operator()(const T& src) const
        {
            return saturate_cast<D>(alpha * src + beta);
        }

        const double alpha, beta;
    };
    
    template<typename T, typename D>
    void cvt_(const DevMem2D& src, const DevMem2D& dst, double alpha, double beta, hipStream_t stream)
    {
        cudaSafeCall( cudaSetDoubleForDevice(&alpha) );
        cudaSafeCall( cudaSetDoubleForDevice(&beta) );
        Convertor<T, D> op(alpha, beta);
        transform((DevMem2D_<T>)src, (DevMem2D_<D>)dst, op, stream);
    }

    void convert_gpu(const DevMem2D& src, int sdepth, const DevMem2D& dst, int ddepth, double alpha, double beta, 
        hipStream_t stream = 0)
    {
        typedef void (*caller_t)(const DevMem2D& src, const DevMem2D& dst, double alpha, double beta, 
            hipStream_t stream);

        static const caller_t tab[8][8] =
        {
            {cvt_<uchar, uchar>, cvt_<uchar, schar>, cvt_<uchar, ushort>, cvt_<uchar, short>,
            cvt_<uchar, int>, cvt_<uchar, float>, cvt_<uchar, double>, 0},

            {cvt_<schar, uchar>, cvt_<schar, schar>, cvt_<schar, ushort>, cvt_<schar, short>,
            cvt_<schar, int>, cvt_<schar, float>, cvt_<schar, double>, 0},

            {cvt_<ushort, uchar>, cvt_<ushort, schar>, cvt_<ushort, ushort>, cvt_<ushort, short>,
            cvt_<ushort, int>, cvt_<ushort, float>, cvt_<ushort, double>, 0},

            {cvt_<short, uchar>, cvt_<short, schar>, cvt_<short, ushort>, cvt_<short, short>,
            cvt_<short, int>, cvt_<short, float>, cvt_<short, double>, 0},

            {cvt_<int, uchar>, cvt_<int, schar>, cvt_<int, ushort>,
            cvt_<int, short>, cvt_<int, int>, cvt_<int, float>, cvt_<int, double>, 0},

            {cvt_<float, uchar>, cvt_<float, schar>, cvt_<float, ushort>,
            cvt_<float, short>, cvt_<float, int>, cvt_<float, float>, cvt_<float, double>, 0},

            {cvt_<double, uchar>, cvt_<double, schar>, cvt_<double, ushort>,
            cvt_<double, short>, cvt_<double, int>, cvt_<double, float>, cvt_<double, double>, 0},

            {0,0,0,0,0,0,0,0}
        };

        caller_t func = tab[sdepth][ddepth];
        if (!func)
            cv::gpu::error("Unsupported convert operation", __FILE__, __LINE__);

        func(src, dst, alpha, beta, stream);
    }
}}}
