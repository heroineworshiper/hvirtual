#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudaarithm.hpp"
#include "opencv2/cudev.hpp"

using namespace cv::cudev;

void cv::cuda::magnitude(InputArray _x, InputArray _y, OutputArray _dst, Stream& stream)
{
    GpuMat x = _x.getGpuMat();
    GpuMat y = _y.getGpuMat();

    CV_DbgAssert( x.depth() == CV_32F );
    CV_DbgAssert( y.type() == x.type() && y.size() == x.size() );

    _dst.create(x.size(), CV_32FC1);
    GpuMat dst = _dst.getGpuMat();

    GpuMat_<float> xc(x.reshape(1));
    GpuMat_<float> yc(y.reshape(1));
    GpuMat_<float> magc(dst.reshape(1));

    gridTransformBinary(xc, yc, magc, magnitude_func<float>(), stream);
}

void cv::cuda::magnitudeSqr(InputArray _x, InputArray _y, OutputArray _dst, Stream& stream)
{
    GpuMat x = _x.getGpuMat();
    GpuMat y = _y.getGpuMat();

    CV_DbgAssert( x.depth() == CV_32F );
    CV_DbgAssert( y.type() == x.type() && y.size() == x.size() );

    _dst.create(x.size(), CV_32FC1);
    GpuMat dst = _dst.getGpuMat();

    GpuMat_<float> xc(x.reshape(1));
    GpuMat_<float> yc(y.reshape(1));
    GpuMat_<float> magc(dst.reshape(1));

    gridTransformBinary(xc, yc, magc, magnitude_sqr_func<float>(), stream);
}

void cv::cuda::phase(InputArray _x, InputArray _y, OutputArray _dst, bool angleInDegrees, Stream& stream)
{
    GpuMat x = _x.getGpuMat();
    GpuMat y = _y.getGpuMat();

    CV_DbgAssert( x.depth() == CV_32F );
    CV_DbgAssert( y.type() == x.type() && y.size() == x.size() );

    _dst.create(x.size(), CV_32FC1);
    GpuMat dst = _dst.getGpuMat();

    GpuMat_<float> xc(x.reshape(1));
    GpuMat_<float> yc(y.reshape(1));
    GpuMat_<float> anglec(dst.reshape(1));

    if (angleInDegrees)
        gridTransformBinary(xc, yc, anglec, direction_func<float, true>(), stream);
    else
        gridTransformBinary(xc, yc, anglec, direction_func<float, false>(), stream);
}

void cv::cuda::cartToPolar(InputArray _x, InputArray _y, OutputArray _mag, OutputArray _angle, bool angleInDegrees, Stream& stream)
{
    GpuMat x = _x.getGpuMat();
    GpuMat y = _y.getGpuMat();

    CV_DbgAssert( x.depth() == CV_32F );
    CV_DbgAssert( y.type() == x.type() && y.size() == x.size() );

    _mag.create(x.size(), CV_32FC1);
    GpuMat mag = _mag.getGpuMat();

    _angle.create(x.size(), CV_32FC1);
    GpuMat angle = _angle.getGpuMat();

    GpuMat_<float> xc(x.reshape(1));
    GpuMat_<float> yc(y.reshape(1));
    GpuMat_<float> magc(mag.reshape(1));
    GpuMat_<float> anglec(angle.reshape(1));

    if (angleInDegrees)
    {
        gridTransformTuple(zipPtr(xc, yc),
                           tie(magc, anglec),
                           make_tuple(
                               binaryTupleAdapter<0, 1>(magnitude_func<float>()),
                               binaryTupleAdapter<0, 1>(direction_func<float, true>())),
                           stream);
    }
    else
    {
        gridTransformTuple(zipPtr(xc, yc),
                           tie(magc, anglec),
                           make_tuple(
                               binaryTupleAdapter<0, 1>(magnitude_func<float>()),
                               binaryTupleAdapter<0, 1>(direction_func<float, false>())),
                           stream);
    }
}

namespace
{
    template <bool useMag>
    __global__ void polarToCartImpl(const GlobPtr<float> mag, const GlobPtr<float> angle, GlobPtr<float> xmat, GlobPtr<float> ymat, const float scale, const int rows, const int cols)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x >= cols || y >= rows)
            return;

        const float mag_val = useMag ? mag(y, x) : 1.0f;
        const float angle_val = angle(y, x);

        float sin_a, cos_a;
        ::sincosf(scale * angle_val, &sin_a, &cos_a);

        xmat(y, x) = mag_val * cos_a;
        ymat(y, x) = mag_val * sin_a;
    }
}

void cv::cuda::polarToCart(InputArray _mag, InputArray _angle, OutputArray _x, OutputArray _y, bool angleInDegrees, Stream& _stream)
{
    GpuMat mag = _mag.getGpuMat();
    GpuMat angle = _angle.getGpuMat();

    CV_DbgAssert( angle.depth() == CV_32F );
    CV_DbgAssert( mag.empty() || (mag.type() == angle.type() && mag.size() == angle.size()) );

    _x.create(angle.size(), CV_32FC1);
    GpuMat x = _x.getGpuMat();

    _y.create(angle.size(), CV_32FC1);
    GpuMat y = _y.getGpuMat();

    GpuMat_<float> xc(x.reshape(1));
    GpuMat_<float> yc(y.reshape(1));
    GpuMat_<float> magc(mag.reshape(1));
    GpuMat_<float> anglec(angle.reshape(1));

    const dim3 block(32, 8);
    const dim3 grid(divUp(anglec.cols, block.x), divUp(anglec.rows, block.y));

    const float scale = angleInDegrees ? (CV_PI_F / 180.0f) : 1.0f;

    hipStream_t stream = StreamAccessor::getStream(_stream);

    if (magc.empty())
        polarToCartImpl<false><<<grid, block, 0, stream>>>(shrinkPtr(magc), shrinkPtr(anglec), shrinkPtr(xc), shrinkPtr(yc), scale, anglec.rows, anglec.cols);
    else
        polarToCartImpl<true><<<grid, block, 0, stream>>>(shrinkPtr(magc), shrinkPtr(anglec), shrinkPtr(xc), shrinkPtr(yc), scale, anglec.rows, anglec.cols);

    CV_CUDEV_SAFE_CALL( hipGetLastError() );

    if (stream == 0)
        CV_CUDEV_SAFE_CALL( hipDeviceSynchronize() );
}

#endif
