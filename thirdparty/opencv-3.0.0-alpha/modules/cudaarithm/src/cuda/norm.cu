#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudaarithm.hpp"
#include "opencv2/cudev.hpp"

using namespace cv::cudev;

namespace
{
    double normDiffInf(const GpuMat& _src1, const GpuMat& _src2, GpuMat& _buf)
    {
        const GpuMat_<uchar>& src1 = (const GpuMat_<uchar>&) _src1;
        const GpuMat_<uchar>& src2 = (const GpuMat_<uchar>&) _src2;
        GpuMat_<int>& buf = (GpuMat_<int>&) _buf;

        gridFindMinMaxVal(abs_(cvt_<int>(src1) - cvt_<int>(src2)), buf);

        int data[2];
        buf.download(cv::Mat(1, 2, buf.type(), data));

        return data[1];
    }

    double normDiffL1(const GpuMat& _src1, const GpuMat& _src2, GpuMat& _buf)
    {
        const GpuMat_<uchar>& src1 = (const GpuMat_<uchar>&) _src1;
        const GpuMat_<uchar>& src2 = (const GpuMat_<uchar>&) _src2;
        GpuMat_<int>& buf = (GpuMat_<int>&) _buf;

        gridCalcSum(abs_(cvt_<int>(src1) - cvt_<int>(src2)), buf);

        int data;
        buf.download(cv::Mat(1, 1, buf.type(), &data));

        return data;
    }

    double normDiffL2(const GpuMat& _src1, const GpuMat& _src2, GpuMat& _buf)
    {
        const GpuMat_<uchar>& src1 = (const GpuMat_<uchar>&) _src1;
        const GpuMat_<uchar>& src2 = (const GpuMat_<uchar>&) _src2;
        GpuMat_<double>& buf = (GpuMat_<double>&) _buf;

        gridCalcSum(sqr_(cvt_<double>(src1) - cvt_<double>(src2)), buf);

        double data;
        buf.download(cv::Mat(1, 1, buf.type(), &data));

        return std::sqrt(data);
    }
}

double cv::cuda::norm(InputArray _src1, InputArray _src2, GpuMat& buf, int normType)
{
    typedef double (*func_t)(const GpuMat& _src1, const GpuMat& _src2, GpuMat& _buf);
    static const func_t funcs[] =
    {
        0, normDiffInf, normDiffL1, 0, normDiffL2
    };

    GpuMat src1 = _src1.getGpuMat();
    GpuMat src2 = _src2.getGpuMat();

    CV_Assert( src1.type() == CV_8UC1 );
    CV_Assert( src1.size() == src2.size() && src1.type() == src2.type() );
    CV_Assert( normType == NORM_INF || normType == NORM_L1 || normType == NORM_L2 );

    return funcs[normType](src1, src2, buf);
}

#endif
